/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <chrono>
#include <cstdint>
#include <fstream>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <string>
#include <unordered_set>
#include <vector>

#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/core/host_mdarray.hpp>
#include <raft/core/mdspan_types.hpp>
#include <raft/util/cudart_utils.hpp>

#include <cuvs/neighbors/cagra.hpp>

#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

/**
 * @brief Reads vectors from a file in fvecs format.
 *
 * The fvecs format consists of a 4-byte integer (vector dimension),
 * followed by vectors stored as 4-byte floats.
 *
 * @param handle RAFT device resources handle
 * @param file_path Path to the fvecs file
 * @param dim Output parameter for vector dimension
 * @param n_rows Output parameter for the number of vectors read
 * @param max_rows_to_read The maximum number of vectors to read from the file. -1 means all.
 * @return raft::device_matrix<float, int64_t> A device matrix containing the data
 */
raft::device_matrix<float, int64_t> read_fvecs(raft::device_resources const& handle,
                                               const std::string& file_path,
                                               int64_t& dim,
                                               int64_t& n_rows,
                                               int64_t max_rows_to_read = -1)
{
  std::ifstream is(file_path, std::ios::binary);
  if (!is.is_open()) { throw std::runtime_error("Could not open file: " + file_path); }

  // Read dimension
  int d;
  is.read(reinterpret_cast<char*>(&d), sizeof(int));
  dim = static_cast<int64_t>(d);

  // Get file size to calculate total number of vectors in file
  is.seekg(0, std::ios::end);
  size_t file_size = is.tellg();
  is.seekg(0, std::ios::beg);
  size_t vector_size_bytes = (1 * sizeof(int)) + (dim * sizeof(float));
  int64_t total_rows_in_file = file_size / vector_size_bytes;

  // Determine how many rows to read
  n_rows = total_rows_in_file;
  if (max_rows_to_read > 0 && max_rows_to_read < total_rows_in_file) {
    n_rows = max_rows_to_read;
  }

  // Read data into host memory
  std::cout << "Loading " << n_rows << " vectors of dimension " << dim << " from " << file_path
            << std::endl;
  std::vector<float> host_data(n_rows * dim);
  for (int64_t i = 0; i < n_rows; ++i) {
    is.seekg(sizeof(int), std::ios::cur);  // Skip dimension
    is.read(reinterpret_cast<char*>(host_data.data() + i * dim), dim * sizeof(float));
  }
  is.close();

  // Copy to device
  auto device_mat = raft::make_device_matrix<float, int64_t>(handle, n_rows, dim);
  raft::update_device(device_mat.data_handle(), host_data.data(), n_rows * dim, handle.get_stream());
  handle.sync_stream();
  return device_mat;
}

/**
 * @brief Reads vectors from a file in ivecs format.
 *
 * The ivecs format is similar to fvecs but with integer data.
 *
 * @param handle RAFT device resources handle
 * @param file_path Path to the ivecs file
 * @param dim Output parameter for vector dimension
 * @param n_rows Output parameter for the number of vectors
 * @return raft::device_matrix<uint32_t, int64_t> A device matrix containing the data
 */
raft::device_matrix<uint32_t, int64_t> read_ivecs(raft::device_resources const& handle,
                                                  const std::string& file_path,
                                                  int64_t& dim,
                                                  int64_t& n_rows)
{
  std::ifstream is(file_path, std::ios::binary);
  if (!is.is_open()) { throw std::runtime_error("Could not open file: " + file_path); }

  // Read dimension
  int d;
  is.read(reinterpret_cast<char*>(&d), sizeof(int));
  dim = static_cast<int64_t>(d);

  // Get file size to calculate number of vectors
  is.seekg(0, std::ios::end);
  size_t file_size = is.tellg();
  is.seekg(0, std::ios::beg);
  size_t vector_size_bytes = (1 * sizeof(int)) + (dim * sizeof(int));
  n_rows                 = file_size / vector_size_bytes;

  // Read data into host memory
  std::cout << "Loading " << n_rows << " vectors of dimension " << dim << " from " << file_path
            << std::endl;
  std::vector<uint32_t> host_data(n_rows * dim);
  for (int64_t i = 0; i < n_rows; ++i) {
    is.seekg(sizeof(int), std::ios::cur);  // Skip dimension
    is.read(reinterpret_cast<char*>(host_data.data() + i * dim), dim * sizeof(uint32_t));
  }
  is.close();

  // Copy to device
  auto device_mat = raft::make_device_matrix<uint32_t, int64_t>(handle, n_rows, dim);
  raft::update_device(device_mat.data_handle(), host_data.data(), n_rows * dim, handle.get_stream());
  handle.sync_stream();
  return device_mat;
}

/**
 * @brief Calculates the recall@K metric.
 *
 * Recall is the fraction of true nearest neighbors that are found by the search.
 *
 * @param handle RAFT device resources handle
 * @param neighbors The neighbor indices found by the search algorithm
 * @param ground_truth The ground truth neighbor indices
 */
void calculate_recall(raft::device_resources const& handle,
                      raft::device_matrix_view<const uint32_t, int64_t> neighbors,
                      raft::device_matrix_view<const uint32_t, int64_t> ground_truth)
{
  handle.sync_stream();
  int64_t n_queries = neighbors.extent(0);
  int64_t topk      = neighbors.extent(1);
  int64_t gt_k      = ground_truth.extent(1);

  // Copy data to host for calculation
  auto host_neighbors    = raft::make_host_matrix<uint32_t, int64_t>(n_queries, topk);
  auto host_ground_truth = raft::make_host_matrix<uint32_t, int64_t>(n_queries, gt_k);
  raft::update_host(
    host_neighbors.data_handle(), neighbors.data_handle(), n_queries * topk, handle.get_stream());
  raft::update_host(host_ground_truth.data_handle(),
                    ground_truth.data_handle(),
                    n_queries * gt_k,
                    handle.get_stream());
  handle.sync_stream();

  int total_found = 0;
  for (int64_t i = 0; i < n_queries; ++i) {
    std::unordered_set<uint32_t> gt_set;
    for (int64_t j = 0; j < topk; ++j) {
      gt_set.insert(host_ground_truth(i, j));
    }

    for (int64_t j = 0; j < topk; ++j) {
      if (gt_set.count(host_neighbors(i, j))) { total_found++; }
    }
  }

  float recall = static_cast<float>(total_found) / (static_cast<float>(n_queries) * topk);
  std::cout << "Recall@" << topk << ": " << recall << std::endl;
}

/**
 * @brief Build CAGRA index, search, and evaluate on SIFT1B dataset.
 *
 * @param dev_resources RAFT device resources handle
 * @param dataset The training dataset (e.g., sift1b_base)
 * @param queries The query vectors (e.g., sift1b_query)
 * @param ground_truth The ground truth nearest neighbors for the queries
 */
void cagra_on_sift1b(raft::device_resources const& dev_resources,
                     raft::device_matrix_view<const float, int64_t> dataset,
                     raft::device_matrix_view<const float, int64_t> queries,
                     raft::device_matrix_view<const uint32_t, int64_t> ground_truth)
{
  using namespace cuvs::neighbors;

  // We will search for the top-K neighbors.
  // The ground truth file might contain more neighbors than we want to search for (e.g., top-100),
  // so we'll use a smaller `topk` for the actual search and recall calculation.
  int64_t topk      = 10;
  int64_t n_queries = queries.extent(0);

  // Create output arrays on the device
  auto neighbors = raft::make_device_matrix<uint32_t>(dev_resources, n_queries, topk);
  auto distances = raft::make_device_matrix<float>(dev_resources, n_queries, topk);

  // Configure CAGRA index parameters
  cagra::index_params index_params;
  index_params.intermediate_graph_degree = 128;
  index_params.graph_degree              = 64;

  std::cout << "Building CAGRA index (search graph)..." << std::endl;
  auto start_build = std::chrono::high_resolution_clock::now();
  auto index       = cagra::build(dev_resources, index_params, dataset);
  dev_resources.sync_stream();  // Ensure build is complete before stopping timer
  auto stop_build = std::chrono::high_resolution_clock::now();
  auto build_duration =
    std::chrono::duration_cast<std::chrono::milliseconds>(stop_build - start_build);
  std::cout << "Index construction time: " << build_duration.count() << " ms" << std::endl;

  std::cout << "CAGRA index has " << index.size() << " vectors" << std::endl;
  std::cout << "CAGRA graph has degree " << index.graph_degree() << ", graph size ["
            << index.graph().extent(0) << ", " << index.graph().extent(1) << "]" << std::endl;

  // Configure search parameters
  cagra::search_params search_params;
  search_params.max_queries = 1000;  // Batch size for search

  // Search K nearest neighbors
  std::cout << "Searching for " << topk << " nearest neighbors..." << std::endl;
  auto start_search = std::chrono::high_resolution_clock::now();
  cagra::search(dev_resources, search_params, index, queries, neighbors.view(), distances.view());
  dev_resources.sync_stream();  // Ensure search is complete before stopping timer
  auto stop_search = std::chrono::high_resolution_clock::now();
  auto search_duration =
    std::chrono::duration_cast<std::chrono::milliseconds>(stop_search - start_search);

  std::cout << "Search time: " << search_duration.count() << " ms" << std::endl;
  std::cout << "Search throughput: "
            << static_cast<double>(n_queries) / (search_duration.count() / 1000.0) << " QPS"
            << std::endl;

  // Evaluate the results
  std::cout << "Calculating recall..." << std::endl;
  
  // **FIXED LINE:** Create a sub-view of the ground truth to match the number of neighbors we searched for (topk).
  // This is the correct way to create a sub-view for this RAFT version.
  auto ground_truth_view = raft::make_device_matrix_view(
    ground_truth.data_handle(), ground_truth.extent(0), topk);
    
  calculate_recall(dev_resources, neighbors.view(), ground_truth_view);
}

int main(int argc, char** argv)
{
  if (argc < 4 || argc > 5) {
    std::cerr << "Usage: " << argv[0]
              << " <base_fvecs_path> <query_fvecs_path> <groundtruth_ivecs_path> "
                 "[num_dataset_vectors]"
              << std::endl;
    std::cerr << "  [num_dataset_vectors] (optional): Number of vectors to use from the base file. "
                 "If not specified, all vectors are used."
              << std::endl;
    std::cerr << "Example (1M vectors): " << argv[0]
              << " sift1b_base.fvecs sift1b_query.fvecs sift1b_groundtruth.ivecs 1000000"
              << std::endl;
    return 1;
  }

  std::string base_path                  = argv[1];
  std::string query_path                 = argv[2];
  std::string gt_path                    = argv[3];
  long long num_dataset_vectors_to_use = -1;
  if (argc == 5) { num_dataset_vectors_to_use = std::stoll(argv[4]); }

  raft::device_resources dev_resources;

  // With ample system RAM, we can remove the RMM pool memory resource.
  // This allows RMM to use the default hipMalloc/hipFree for memory management,
  // which can be simpler and avoid pool-size limitations on high-memory systems.
  //
  // rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> pool_mr(
  //   rmm::mr::get_current_device_resource(), 4ull * 1024 * 1024 * 1024);
  // rmm::mr::set_current_device_resource(&pool_mr);

  // Load datasets from files
  int64_t n_samples, n_dim_samples, n_queries, n_dim_queries, n_gt, n_dim_gt;
  auto dataset =
    read_fvecs(dev_resources, base_path, n_dim_samples, n_samples, num_dataset_vectors_to_use);
  auto queries      = read_fvecs(dev_resources, query_path, n_dim_queries, n_queries);
  auto ground_truth = read_ivecs(dev_resources, gt_path, n_dim_gt, n_gt);

  if (n_dim_samples != n_dim_queries) {
    std::cerr << "Error: Dataset and query dimensions do not match (" << n_dim_samples << " vs "
              << n_dim_queries << ")" << std::endl;
    return 1;
  }
  if (n_queries != n_gt) {
    std::cerr << "Error: Number of queries and ground truth entries do not match (" << n_queries
              << " vs " << n_gt << ")" << std::endl;
    return 1;
  }

  // Run the build, search, and evaluation
  cagra_on_sift1b(dev_resources,
                  raft::make_const_mdspan(dataset.view()),
                  raft::make_const_mdspan(queries.view()),
                  raft::make_const_mdspan(ground_truth.view()));

  return 0;
}
